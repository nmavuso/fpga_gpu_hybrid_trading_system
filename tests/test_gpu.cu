// test_gpu.cu
// Test GPU kernels in isolation.


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

// Declarations of the GPU kernels
extern "C" void runPrediction(const float* d_input, float* d_output, float weight, int n, hipStream_t stream);
extern "C" void runRiskCheck(const float* d_positions, float* d_riskFlags, float threshold, int n, hipStream_t stream);
extern "C" void runAdvancedStrategy(const float* d_inData, float* d_outSignals, int dataSize, hipStream_t stream);

int main() {
    std::cout << "[TEST] GPU Test Starting..." << std::endl;

    // Basic test data
    const int N = 10;
    std::vector<float> input(N, 100.0f);
    std::vector<float> output(N, 0.0f);
    std::vector<float> flags(N, 0.0f);

    float *dInput = nullptr, *dOutput = nullptr, *dFlags = nullptr;
    hipMalloc(&dInput,  N * sizeof(float));
    hipMalloc(&dOutput, N * sizeof(float));
    hipMalloc(&dFlags,  N * sizeof(float));

    hipMemcpy(dInput, input.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipStream_t s1, s2, s3;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipStreamCreate(&s3);

    runPrediction(dInput, dOutput, 1.05f, N, s1);
    runRiskCheck(dInput, dFlags, 120.0f, N, s2);
    runAdvancedStrategy(dInput, dOutput, N, s3);

    hipStreamSynchronize(s1);
    hipStreamSynchronize(s2);
    hipStreamSynchronize(s3);

    hipMemcpy(output.data(), dOutput, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(flags.data(),  dFlags,  N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dInput);
    hipFree(dOutput);
    hipFree(dFlags);

    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipStreamDestroy(s3);

    // Validate
    bool pass = true;
    // Just a simple check to see if we got some output
    // (In real tests, you'd compare with expected results.)
    if (std::fabs(output[0]) < 0.0001f) {
        pass = false;
    }

    if (pass) std::cout << "[TEST] GPU Test Passed!" << std::endl;
    else      std::cout << "[TEST] GPU Test Failed!" << std::endl;

    return pass ? 0 : 1;
}
